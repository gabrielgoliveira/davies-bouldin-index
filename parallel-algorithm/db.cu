
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <map>
#include <fstream>

#define DEBUG 1
#define BLOCK_SIZE 128
#define BASE_PATH "/home/gabriel/Desktop/ufg/tcc/dunn-index/"
#define NF 64

using namespace std;

char paths_datasets[][100] = {
    "../datasets/digits_k10_f64_1797.dat", 
    "../datasets/iris_k3_f4_150.dat"
};

int get_nblocks(int size_cluster) {
    return (size_cluster + BLOCK_SIZE - 1) / BLOCK_SIZE;
}


char* get_path_dataset(int dataset_id) {
    return paths_datasets[dataset_id];
}

// Função para imprimir uma matriz
void printMatrix(float **matrix, int n_rows, int n_columns) {
    for (int i = 0; i < n_rows; i++) {
        for (int j = 0; j < n_columns; j++) {
            float val = matrix[i][j];
            cout << val << " ";
        }
        cout << endl;
    }
}

float** malloc_matrix(int n_rows, int n_columns) {
    float** matrix = (float**) malloc(n_rows*sizeof(float*));

    for (int i = 0; i < n_rows; i++) {
        matrix[i] = (float*)malloc(n_columns * sizeof(float));
    }

    return matrix; 
}

void free_matrix(float** matrix, int n_rows, int n_columns) {

    for (int i = 0; i < n_rows; ++i) {
        free(matrix[i]);
    }

    // Liberando memória para o array de ponteiros para linhas
    free(matrix);
}

/* ===> FUNÇÕES DE KERNEL */

// Função de kernel para imprimir a matriz na GPU
__global__ void cuda_print_matrix(float* d_matrix, int n_rows, int n_columns) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < n_rows && idy < n_columns && idx == 179) {
        printf("matrix[%d][%d] = %f\n", idx, idy, d_matrix[idx * n_columns + idy]);
    }
}

float* cuda_malloc_matrix(int n_rows, int n_columns) {
    float* d_matrix;
    hipMalloc(&d_matrix, n_rows * n_columns * sizeof(float));
    return d_matrix;
}

void cuda_copy_matrix_host_to_device(float* d_matrix, float** h_matrix, int n_rows, int n_columns) {
    for (int i = 0; i < n_rows; i++) {  
        hipMemcpy(d_matrix + i * n_columns, h_matrix[i], n_columns * sizeof(float), hipMemcpyHostToDevice);
    }

    return ;
}

__global__ void d_reduce_points(float *d_cluster, float *d_centroid_tmp, int size, int n_feat) {
    //extern __shared__ float s_centroid[];
     __shared__ float s_centroid[BLOCK_SIZE * NF];

    int tid = threadIdx.x;
    int i = (blockIdx.x * blockDim.x) + tid;

    for (int d = 0; d < n_feat; d++) {
        s_centroid[tid * n_feat + d] = 0.0;
    }

    __syncthreads();

    if (i < size) {
        // copia a linha do cluster referente a thread para s_centroid
        for (int d = 0; d < n_feat; d++) {
            s_centroid[tid * n_feat + d] = (float ) d_cluster[i*n_feat + d];
        }
    }
    __syncthreads();

    int p = blockDim.x / 2; // numero de threads dentro do bloco dividido por 2
    while (p != 0) {
        if (tid < p) {
            for (int d = 0; d < n_feat; d++) {
            s_centroid[tid*n_feat+d] = s_centroid[tid*n_feat+d] + s_centroid[(tid+p)*n_feat+d];
        }
        }
        __syncthreads();
        p = p/2;
    }

    // Thread zero of each block moves the local result to the global memory
    if (tid == 0) {
        for (int d = 0; d < n_feat; d++) {
            d_centroid_tmp[blockIdx.x * n_feat + d] = (float )s_centroid[d];
        }
    }

    return ;
}

void cuda_verifica_erros(hipError_t error) {
    if(error != hipSuccess) { 
        printf("CUDA error: %s\n", hipGetErrorString(error)); 
        exit(-1); 
    }
}

int main() {

    int n_clusters, n_feat, count = 0;
    vector<int>       size_clusters;
    map<int, float*>  centroids;
    map<int, float**> clusters;
    map<int, float*>  d_clusters;         // Enderecos dos clusters alocados na device (gpu)
    map<int, float*>  d_partial_centroid; // DEVICE: centroides parciais obtidos por meio de redução

    clock_t start, stop;
    double running_time;

    char *path_dataset = get_path_dataset(0);
    ifstream dataset(path_dataset);

    /*
        ==> STEP 1: LER O ARQUIVO
    */
    
    dataset >> n_clusters >> n_feat; // primeira linha do arquivo

    cout<<"================= INFOS DATASET LIDO ========================\n";
    cout<<"Qtd. clusters: "<<n_clusters<<" Qtd. Features: "<<n_feat<<endl;
    cout<<"=============================================================\n";

    
    for (int i = 0; i < n_clusters; i++) {
        // segunda linha do arquivo (lê o tamanho dos clusters)
        int size_cluster = 0;
        dataset >> size_cluster;
        size_clusters.push_back(size_cluster);
    }

    
    for (int i = 0; i < size_clusters.size(); i++) {
        // percorrer o arquivo em relação a cada cluster
        int size_current_cluster = size_clusters[i];
        float** current_cluster = malloc_matrix(size_current_cluster, n_feat);
           
        for (int j = 0; j < size_current_cluster; j++) { // le o cluster
            for(int k = 0; k < n_feat; k++) { // le a linha do arquivo
                float value;
                dataset >> value;
                current_cluster[j][k] = value;
            }
        }

        clusters.insert(pair<int, float**>(i, current_cluster));
    }

    if(DEBUG == 1) {
        count = 0;
        for (map<int, float**>::iterator it = clusters.begin(); it != clusters.end(); ++it) {
            // printMatrix(it->second, size_clusters[count], n_feat);
            count++;
        }
    }
    
    /*
        ==> STEP 2: ALOCA MEMORIA NA GPU E COPIA DADOS PARA A MEMORIA DA GPU
    */

    for (int i = 0; i < n_clusters; i++) {
        float* d_cluster, *d_centroid_temp;
        int size_current_cluster = size_clusters[i];

        // aloca memoria na gpu
        d_cluster = cuda_malloc_matrix(size_current_cluster, n_feat);
        d_clusters.insert(pair<int, float*>(i, d_cluster));
 

        // copia matriz em memoria para a GPU
        float **h_cluster = clusters[i];
        cuda_copy_matrix_host_to_device(d_cluster, h_cluster, size_current_cluster, n_feat);
    }

    if(DEBUG == 1) {
        printf("Memoria alocada na GPU e dados copiados !!\n");
    }

/*
    float *last_cluster = d_clusters[9];
    int size_last_cluster = size_clusters[9];

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((size_last_cluster + threadsPerBlock.x - 1) / threadsPerBlock.x,  (n_feat + threadsPerBlock.y - 1) / threadsPerBlock.y);

    cuda_print_matrix<<<numBlocks,  threadsPerBlock>>>(last_cluster, size_last_cluster, n_feat);
    cudaDeviceSynchronize();
*/

    /*
        ==> STEP 3: Calcular o centroide
    */

    float *d_centroid_tmp;
    for (int i = 0; i < n_clusters; i++) {
        hipDeviceSynchronize();
        int cluster_size = size_clusters[i];
        int nblocks = get_nblocks(cluster_size);
        float *d_current_cluster = d_clusters[i];
        float *h_reduce = (float*) malloc(sizeof(float)*nblocks*n_feat);
        
        d_centroid_tmp = cuda_malloc_matrix(nblocks, n_feat);
        d_reduce_points <<<nblocks, BLOCK_SIZE>>>(
            d_current_cluster, // Ponteiro do cluster no device
            d_centroid_tmp,    // reducao dos pontos em relacao aos blocos
            cluster_size,      // tamanho do cluster
            n_feat            // numero de features
        );

        hipError_t error = hipGetLastError();
        cuda_verifica_erros(error);
        
        hipDeviceSynchronize();
        hipMemcpy(h_reduce, d_centroid_tmp, nblocks*n_feat*sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        float *centroid_current_cluster = (float*) malloc(sizeof(float)*n_feat);

        for (int i = 0; i < n_feat; i++) {
            float sum = 0.0;
            for (int j = 0; j < nblocks; j++) {
                int current_index = j * n_feat + i;
                sum += h_reduce[current_index];
            }
            centroid_current_cluster[i] = sum/cluster_size;
            // printf("%f ", sum);
        }

        if(DEBUG == 1) {
            cout << "\n ===> Centroid do cluster " << i << " : ";

            for (int j = 0; j < n_feat; j++) {
                cout << centroid_current_cluster[j] << " ";
            }
            cout << endl;
        }

    }



    // libera memoria
    count = 0;
    for (map<int, float**>::iterator it = clusters.begin(); it != clusters.end(); ++it) {
        free_matrix(it->second, size_clusters[count], n_feat);
        count++;
    }

    return 0;    
}